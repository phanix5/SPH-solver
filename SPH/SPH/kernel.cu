#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <string>
#include <algorithm>
#include "lib\glew\glew.h"
#include "lib\freeglut\freeglut.h"
#include <time.h>
#include <ctime>
#include "config.h"
#include "Parameters.h"

using namespace std;



void simulation();

int main() {
	glutInit(&__argc, __argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowPosition(920, 00);
	glutInitWindowSize(1000, 1000);
	glutCreateWindow("Smoothed Particle Hydrodynamics");
	glutDisplayFunc(simulation);
	glutMainLoop();
	getchar();
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////
/////// FUNCTIONS
int render_scene(int ntotal, double *x1, double *x2, double* mass, int* fluid_code, int* wall, int* state, int* test_array, int* ghost, double** temp, vector<int> &marker_vector, double part_spacing);
int render_scene_mass(int ntotal, double *x1, double *x2, double* mass, int *wall, int *state, int *ghost, double** temp, double part_spacing);
void dump_data(int ntotal, double *x1, double *x2, double *v1, double *v2, double *rho, double *ref_rho, double *pressure, double *mass, double *eta, double *hsml, double *temp, double *energy, double *therm_cond, double *sp_heat, int *fluid_code, int *state, int *ghost, int* ghost_state, int* wall, int* boundry, int* Interface, double *v1_old, double *v2_old, double *energy_old, double *rho_old, double *x1_h, double *x2_h, double *v1_h, double *v2_h, double *rho_h, double *ref_rho_h, double *pressure_h, double *mass_h, double *eta_h, double *hsml_h, double *temp_h, double *energy_h, double *therm_cond_h, double *sp_heat_h, int *fluid_code_h, int *state_h, int *ghost_h, int* ghost_state_h, int* wall_h, int* boundry_h, int* Interface_h, double *v1_old_h, double *v2_old_h, double *energy_old_h, double *rho_old_h, int arraySize, double length, double part_spacing, int index, pair <int, double> solid_line);
double initial_total_mass_util(int ntotal, double *mass_h, int *wall_h);
void write_array(int *arr, int len, string name);
void write_array(double *arr, int len, string name);
void force_comparison_util(double *dv1dt, double *dv2dt, double *dS1dt, double *dS2dt, double *exdv1dt, double *exdv2dt, int len);

// Function to find the maximum smoothing length required in the calculation of timestep.
__global__ void max_hsml(int ntotal, double *hsml, double *hsml_max, int *fluidcode)
{
	hsml_max[0] = hsml[1];
	for (int i = 1; i <= ntotal; i++) if (hsml[i] > *hsml_max && fluidcode[i] != -1)hsml_max[0] = hsml[i];
}

// A voxel search is implemented to find the neighbors of particles. Each particle is assigned to a square of side = smoothing length based on its position
__global__ void index_particles(int ntotal, int *bucket_index, double *x1, double *x2, double length, double kern_constant, double *hsml_max, int *nvoxel_length)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > 0 && idx <= ntotal)
	{

		nvoxel_length[0] = (int)(length / (kern_constant*hsml_max[0]) + 1);
		bucket_index[idx] = (x1[idx]) / (kern_constant*hsml_max[0]) + nvoxel_length[0] * (int)((x2[idx]) / (kern_constant*hsml_max[0]));

	}
}

__global__ void equation_of_state(int ntotal, double *pressure, double *rho, double *ref_rho, int* fluidcode, int* Interface, double p0, double gamma, double background_pressure, double c)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > 0 && idx <= ntotal)
	{
		if (fluidcode[idx] == 1)pressure[idx] = p0*(pow(rho[idx] / ref_rho[idx], gamma) - 1) + background_pressure;  //Simulating Free Surface flows Monaghan, Applied Mathematics
																													 //if (fluidcode[idx] == 2)pressure[idx] = c*c*(pow(rho[idx] / ref_rho[idx], 1.4) - 1) + 0.003;
																													 // pressure[idx]= 0.2*c*c*rho[idx];


		if (fluidcode[idx] > 3)pressure[idx] = 0;
	}
}

__global__ void compute_kernel(int ntotal, int *bucket_index, int *nvoxel_length, double *x1, double *x2, double *hsml, int *neighbors, int *neighbors_count, double *w, double *dwdx, double *dwdy, int *ghost, int *ghost_state, int *wall, int *fluid_code, int *Interface, double kern_constant, double part_spacing, int max_neighbors)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	double dx, dy, r;
	int index;
	neighbors_count[idx] = 0;
	Interface[idx] = 0;
	for (int i = 1; i <= ntotal; i++)
	{
		if ((bucket_index[i] == bucket_index[idx] || bucket_index[idx] == bucket_index[i] + 1 || bucket_index[idx] == bucket_index[i] - 1 || bucket_index[idx] == bucket_index[i] + nvoxel_length[0] || bucket_index[idx] == bucket_index[i] + nvoxel_length[0] - 1 || bucket_index[idx] == bucket_index[i] + nvoxel_length[0] + 1 || bucket_index[idx] == bucket_index[i] - nvoxel_length[0] - 1 || bucket_index[idx] == bucket_index[i] - nvoxel_length[0] + 1 || bucket_index[idx] == bucket_index[i] - nvoxel_length[0]) && i != idx && !wall[idx] && fluid_code[idx] != -1 && fluid_code[i] != -1 && idx>0)
		{
			dx = x1[idx] - x1[i];
			dy = x2[idx] - x2[i];
			r = dx*dx + dy*dy;
			if (r <= kern_constant*kern_constant*(hsml[i] + hsml[idx])*(hsml[i] + hsml[idx]) / 4)
			{
				if (neighbors_count[idx] == max_neighbors)
				{
					printf("\n\nFATAL!! too many neighbhors for the particle #%d\n\n", idx);
					return;
				}
				neighbors_count[idx]++;
				index = idx*max_neighbors + neighbors_count[idx];

				neighbors[index] = i;


				//Tag all non-ghost particles close to interface
				if (ghost[i] && !ghost[idx] && fluid_code[idx] == 1 && !wall[idx])Interface[idx] = 1;

				/////////////////////////////////////////////////////////////////////////////////////////////
				//////////////// KERNEL COMPUTATION
				double hsml_avg = (hsml[i] + hsml[idx]) / 2;
				r = sqrt(r) / hsml_avg;
				double factor = 15.0 / (7.0 * hsml_avg * hsml_avg * 3.14159265358979323846);
				if (r >= 0 && r <= 1.0)
				{
					w[index] = factor * (2.0 / 3.0 - (r * r) + (r * r * r) / 2.0);
					dwdx[index] = factor * (-2.0 + 1.5 * r) / (hsml_avg*hsml_avg) * dx;    // CHECK THIS FUNCTION!!!
					dwdy[index] = factor * (-2.0 + 1.5 * r) / (hsml_avg*hsml_avg) * dy;
				}
				else if (r > 1.0 && r <= 2.0)
				{
					w[index] = factor * (1.0 / 6.0) * (2.0 - r) * (2.0 - r) * (2.0 - r);
					dwdx[index] = -factor * 0.5* (2.0 - r) * (2.0 - r) / hsml_avg * (dx / (r*hsml_avg));
					dwdy[index] = -factor * 0.5* (2.0 - r) * (2.0 - r) / hsml_avg * (dy / (r*hsml_avg));
				}
				else
				{
					w[index] = 0;
					dwdx[index] = 0;
					dwdy[index] = 0;
				}
				/*double factor = 7 / (478 * hsml_avg*hsml_avg*3.14159265358979323846);       //MOrris JOURNAL OF COMPUTATIONAL PHYSICS 136,214–226(1997) Reason for using quintic spline given
				if (r >= 0 && r < 1)
				{
					w[index] = factor*(pow(3 - r, 5) - 6 * pow(2 - r, 5) + 15 * pow(1 - r, 5));
					dwdx[index] = factor * 5 * (-pow(3 - r, 4) + 6 * pow(2 - r, 4) - 15 * pow(1 - r, 4))*dx / (r*hsml_avg*hsml_avg);
					dwdy[index] = factor * 5 * (-pow(3 - r, 4) + 6 * pow(2 - r, 4) - 15 * pow(1 - r, 4))*dy / (r*hsml_avg*hsml_avg);
				}
				else if (r >= 1 && r < 2)
				{
					w[index] = factor*(pow(3 - r, 5) - 6 * pow(2 - r, 5));
					dwdx[index] = factor * 5 * (-pow(3 - r, 4) + 6 * pow(2 - r, 4))*dx / (r*hsml_avg*hsml_avg);
					dwdy[index] = factor * 5 * (-pow(3 - r, 4) + 6 * pow(2 - r, 4))*dy / (r*hsml_avg*hsml_avg);
				}
				else if (r >= 2 && r < 3)
				{
					w[index] = factor*(pow(3 - r, 5));
					dwdx[index] = factor * 5 * (-pow(3 - r, 4))*dx / (r*hsml_avg*hsml_avg);
					dwdy[index] = factor * 5 * (-pow(3 - r, 4))*dy / (r*hsml_avg*hsml_avg);
				}
				else
				{
					w[index] = 0;
					dwdx[index] = 0;
					dwdy[index] = 0;
				}*/
				/////////////////////////////////////////////////////////////////////////////////////////////
			}
		}
	}
}

__global__ void compute_derivatives(int ntotal, double *x1, double *x2, double *v1, double *v2, double *rho, double *mass, double *pressure, double *hsml, int *neighbors, int *neighbors_count, double *w, double *dwdx, double *dwdy, double *eta, double *therm_cond, double *temp, double *dv1dt, double *dv2dt, double *dS1dt, double *dS2dt, double *exdv1dt, double *exdv2dt, double *av1, double *av2, double *drhodt, double *dhdt, double *dmgdt, double * dhsmldt, int *wall, int *fluid_code, int *Interface, int *state, int *ghost_state, int *ghost, double length, double epsilon, double gamma, double c, double back_press, int max_neighbors, int kern_constant, double constant_gravity, double S_ll, double S_ls, double part_spacing)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	if (idx > 0 && (idx - 1) / ntotal == 0)
	{
		double dvx, dvy;
		int neighbor_index;
		int index = max_neighbors*idx;
		int idy = idx;
		drhodt[idy] = 0;
		double w_norm = 0;
		//if (state[idy] == LIQUID)
		{

			for (int i = 1; i <= neighbors_count[idy]; i++)
			{
				neighbor_index = neighbors[index + i];
				//////////////////////////////////////////////////////////////////////////////////////////////////
				/////////////// SUMMATION DENSITY
				/*if (state[neighbor_index] == SOLID && !ghost[neighbor_index])
				{
				drhodt[idy] += 1000 * part_spacing*part_spacing*w[index + i];
				w_norm += part_spacing*part_spacing*w[index + i];
				}
				else
				{
				drhodt[idy] += mass[neighbor_index] * w[index + i];
				w_norm += mass[neighbor_index]*w[index + i] / rho[neighbor_index];
				}*/
				//////////////////////////////////////////////////////////////////////////////////////////////////
				/////////////// CONTINUITY DENSITY MULTIPHASE
				dvx = (v1[idy] - v1[neighbor_index]);
				dvy = (v2[idy] - v2[neighbor_index]);
				if (fluid_code[idy] == fluid_code[neighbor_index]) drhodt[idy] += rho[idy] * (dvx*dwdx[index + i] + dvy*dwdy[index + i])*mass[neighbor_index] / rho[neighbor_index];
				//if (fluid_code[idy] == 1 && fluid_code[neighbor_index] == 1) drhodt[idy] += dmgdt[neighbor_index] * w[index + i];

			}
			//w_norm += mass[idy] * 297471206 / rho[idy];
			//drhodt[idy] += 297471206 * mass[idy];
			//if(drhodt[idy]<=995)drhodt[idy] /= w_norm;
		}
	}

	if ((idx - 1) / ntotal == 1)
	{
		double dvx, dvy, dx, dy;
		int neighbor_index;
		int index = max_neighbors*(idx - ntotal);
		int idy = idx - ntotal;
		dv1dt[idy] = 0;
		dv2dt[idy] = 0;

		for (int i = 1; i <= neighbors_count[idy]; i++)
		{
			neighbor_index = neighbors[index + i];
			dvx = (v1[idy] - v1[neighbor_index]);
			dvy = (v2[idy] - v2[neighbor_index]);
			dx = (x1[idx] - x1[neighbor_index]);
			dy = (x2[idx] - x2[neighbor_index]);

			//if (idy == 854)printf("gradients: %.12f neigh: %d\n", (pressure[neighbor_index]),neighbor_index);
			//////////////////////////////////////////////////////////////////////////////////////////////////
			/////////////// MOMENTUM (Morris et. al. low Re flows)
			if (fluid_code[idy])
			{
				dv1dt[idy] -= (pressure[idy] / pow(rho[idy], 2) + pressure[neighbor_index] / pow(rho[neighbor_index], 2))*dwdx[index + i] * mass[neighbor_index];
				dv2dt[idy] -= (pressure[idy] / pow(rho[idy], 2) + pressure[neighbor_index] / pow(rho[neighbor_index], 2))*dwdy[index + i] * mass[neighbor_index];
			}

			//////////////////////////////////////////////////////////////////////////////////////////////////
			////////////// Low Re term from Morris et.al.
			if (fluid_code[idy])
			{
				dv1dt[idy] += (eta[idy] + eta[neighbor_index])*mass[neighbor_index] * dvx*(dx*dwdx[index + i] + dy*dwdy[index + i]) / (rho[idy] * rho[neighbor_index] * (dx*dx + dy*dy));
				dv2dt[idy] += (eta[idy] + eta[neighbor_index])*mass[neighbor_index] * dvy*(dx*dwdx[index + i] + dy*dwdy[index + i]) / (rho[idy] * rho[neighbor_index] * (dx*dx + dy*dy));
			}

		}
		dv2dt[idy] -= constant_gravity;
	}

	if ((idx - 1) / ntotal == 2)
	{
		double dvx, dvy;
		int neighbor_index;
		int index = max_neighbors*(idx - 2 * ntotal);
		int idy = idx - 2 * ntotal;
		av1[idy] = 0;
		av2[idy] = 0;

		for (int i = 1; i <= neighbors_count[idy]; i++)
		{
			neighbor_index = neighbors[index + i];
			dvx = (v1[idy] - v1[neighbor_index]);
			dvy = (v2[idy] - v2[neighbor_index]);
			//////////////////////////////////////////////////////////////////////////////////////////////////
			////////////// XSPH
			if (fluid_code[idy] == fluid_code[neighbor_index])
			{
				av1[idy] -= 2 * epsilon*mass[neighbor_index] * dvx / (rho[idy] + rho[neighbor_index])*w[index + i];
				av2[idy] -= 2 * epsilon*mass[neighbor_index] * dvy / (rho[idy] + rho[neighbor_index])*w[index + i];
			}
		}
	}


	if ((idx - 1) / ntotal == 3)
	{

		int neighbor_index;
		double dx, dy, hsml_avg, r;
		int index = max_neighbors*(idx - 3 * ntotal);
		int idy = idx - 3 * ntotal;
		dhdt[idy] = 0;

		for (int i = 1; i <= neighbors_count[idy]; i++)
		{
			neighbor_index = neighbors[index + i];
			dx = (x1[idy] - x1[neighbor_index]);
			dy = (x2[idy] - x2[neighbor_index]);
			hsml_avg = (hsml[idy] + hsml[neighbor_index]) / 2;
			r = dx*dx + dy*dy;
			////////////////////////////////////////////////////////////////////////////////////////////////////
			///////////// ENERGY EQUATION

			if (fluid_code[idy] == 1 && fluid_code[neighbor_index] == 1)
			{
				double factor = 4 * therm_cond[idy] * therm_cond[neighbor_index] / (therm_cond[idy] + therm_cond[neighbor_index]);
				dhdt[idy] += factor*(temp[idy] - temp[neighbor_index])*mass[neighbor_index] / (rho[neighbor_index]/**rho[idy]*/) * (dx*dwdx[index + i] + dy*dwdy[index + i]) / (r + 0.01*hsml_avg*hsml_avg);

			}
		}
	}
	if ((idx - 1) / ntotal == 4)
	{
		double dvx, dvy;
		int neighbor_index;
		int index = max_neighbors*(idx - 4 * ntotal);
		int idy = idx - 4 * ntotal;
		dhsmldt[idy] = 0;

		for (int i = 1; i <= neighbors_count[idy]; i++)
		{
			neighbor_index = neighbors[index + i];
			dvx = (v1[idy] - v1[neighbor_index]);
			dvy = (v2[idy] - v2[neighbor_index]);
			////////////////////////////////////////////////////////////////////////////////////////////////////
			////////////// SMOOTHING LENGTH EVOLUTION
			//dhsmldt[idy] -= hsml[idy] * mass[neighbor_index] * (dvx *dwdx[index + i] + dvy *dwdy[index + i]) / (rho[neighbor_index] * 2);
		}
	}
	/*if ((idx - 1) / ntotal == 5)
	{
		int neighbor_index;
		double dx, dy, hsml_avg, r;
		int index = max_neighbors*(idx - 5 * ntotal);
		int idy = idx - 5 * ntotal;
		dS1dt[idy] = 0;
		dS2dt[idy] = 0;
		for (int i = 1; i <= neighbors_count[idy]; i++)
		{
			neighbor_index = neighbors[index + i];
			if (fluid_code[idy] == 1 && state[idy] == LIQUID && !ghost[neighbor_index] && !wall[neighbor_index])
			{
				dx = (x1[idy] - x1[neighbor_index]);
				dy = (x2[idy] - x2[neighbor_index]);
				hsml_avg = (hsml[idy] + hsml[neighbor_index]) / 2;
				r = sqrt(dx*dx + dy*dy);
				double factor = S_ll;
				if (state[neighbor_index] != LIQUID)factor = S_ls;
				//Surface Tension
				dS1dt[idy] += factor*cos(r*3.14159265358979323846 / (2 * kern_constant*hsml_avg) + 3.14159265358979323846 / 6)*dx / (r*mass[idy]);
				dS2dt[idy] += factor*cos(r*3.14159265358979323846 / (2 * kern_constant*hsml_avg) + 3.14159265358979323846 / 6)*dy / (r*mass[idy]);
			}
		}
	}*/
	/*else if ((idx - 1) / ntotal == 6)
	{
	double dx, dy, r;
	int neighbor_index;
	int index = max_neighbors*(idx - 6 * ntotal);
	int idy = idx - 6 * ntotal;
	exdv1dt[idy] = 0;
	int min_index = 0;
	double dx_min = 1e30;

	for (int i = 1; i < neighbors_count[idy]; i++)
	{
	neighbor_index = neighbors[index + i];
	if (ghost[neighbor_index] && !ghost[idy] && state[idy] == LIQUID)
	{

	dx = (x1[idy] - x1[neighbor_index]);
	if (abs(dx) < abs(dx_min) && abs(x2[idy] - x2[neighbor_index]) < 0.7*part_spacing)
	{
	dx_min = dx;
	min_index = neighbor_index;
	}
	}
	}


	//////////////////////////////////////////////////////////////////////////////////////////////////
	///////////// EXTERNAL FORCE
	if (min_index)
	{
	double rr0 = 0.5*part_spacing;
	double dd = 0.01;//was 0.01
	r = 0.5*part_spacing - abs(dx_min);
	double f = (pow(rr0 / r, 12) - pow(rr0 / r, 4)) / pow(r, 2);
	//force scaled using mass of the ghost particle

	if(dx_min>0)exdv1dt[idy] -= dd*r*f;
	else exdv1dt[idy] += dd*r*f;
	//exdv2dt[idy] += mass[neighbor_index] / (917 * part_spacing*part_spacing)*dd*dy*f;

	}

	}
	if ((idx - 1) / ntotal == 7)
	{
	double dy, r;
	int neighbor_index;
	int index = max_neighbors*(idx - 7 * ntotal);
	int idy = idx - 7 * ntotal;
	exdv2dt[idy] = 0;
	int min_index = 0;
	double dy_min = 1e30;

	for (int i = 1; i < neighbors_count[idy]; i++)
	{
	neighbor_index = neighbors[index + i];
	if (ghost[neighbor_index] && !ghost[idy] && state[idy] == LIQUID)
	{

	dy = (x2[idy] - x2[neighbor_index]);
	if (abs(dy) < abs(dy_min) && abs(x1[idy] - x1[neighbor_index]) < 0.7*part_spacing)
	{
	dy_min = dy;
	min_index = neighbor_index;
	}
	}
	}


	//////////////////////////////////////////////////////////////////////////////////////////////////
	///////////// EXTERNAL FORCE
	if (min_index)
	{
	double rr0 = 0.35*part_spacing;
	double dd = 0.01;//was 0.01
	r = abs(dy_min);
	double f = (pow(rr0 / r, 12) - pow(rr0 / r, 4)) / pow(r, 2);
	//force scaled using mass of the ghost particle

	if (dy_min > 0)exdv2dt[idy] += dd*r*f;
	else exdv2dt[idy] -= dd*r*f;
	//exdv2dt[idy] += mass[neighbor_index] / (917 * part_spacing*part_spacing)*dd*dy*f;

	}

	}*/
}

__global__ void leapfrog_part_1(int ntotal, double *v1, double *v2, double *mass, double *ref_rho, double *energy, double *dhdt, double *v1_old, double *v2_old, double *rho, double *rho_old, double *energy_old, int *wall, int *fluidcode, int *ghost, double *drhodt, double *dv1dt, double *dv2dt, double *dt)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > 0 && idx <= ntotal)
	{
		if (!wall[idx] && !ghost[idx] && fluidcode[idx] != -1)
		{
			if (ref_rho[idx] != 917)rho_old[idx] = rho[idx];
			if (ref_rho[idx] != 917)rho[idx] += *dt / 2 * drhodt[idx];
			//if (ref_rho[idx] != 917)rho[idx] = drhodt[idx];
			if (ref_rho[idx] != 917)v1_old[idx] = v1[idx];
			if (ref_rho[idx] != 917)v2_old[idx] = v2[idx];
			if (ref_rho[idx] != 917)v1[idx] += *dt / 2 * dv1dt[idx];
			if (ref_rho[idx] != 917)v2[idx] += *dt / 2 * dv2dt[idx];
			energy_old[idx] = energy[idx];
			energy[idx] += *dt / 2 * mass[idx] * dhdt[idx] / rho[idx];
		}
		if (ghost[idx])
		{
			rho_old[idx] = rho[idx];
			rho[idx] += *dt / 2 * drhodt[idx];
			energy_old[idx] = energy[idx];
			energy[idx] += *dt / 2 * mass[idx] * dhdt[idx] / rho[idx];
		}
	}
}

__global__ void leapfrog_part_2(int ntotal, double *v1, double *v2, double *mass, double *ref_rho, double *temp, double *spheat, double *energy, double *dhdt, double *v1_old, double *v2_old, double *av1, double *av2, double *x1, double *x2, double *rho, double *rho_old, double *energy_old, double *hsml, int *wall, int *fluidcode, int* ghost, int* state, double *drhodt, double *dv1dt, double *dv2dt, double *dhsmldt, double *dt, int *high_compression_flag)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx == 0)*high_compression_flag = 0;
	if (idx > 0 && idx <= ntotal)
	{
		if (!wall[idx] && !ghost[idx] && fluidcode[idx] != -1)
		{
			if (ref_rho[idx] != 917)
			{
				rho[idx] = rho_old[idx] + *dt*drhodt[idx];
				//if (ref_rho[idx] != 917)rho[idx] = drhodt[idx];
				if ((rho[idx] - ref_rho[idx]) > 0.03*ref_rho[idx])*high_compression_flag = 1;
			}
			if (ref_rho[idx] != 917)v1[idx] = v1_old[idx] + *dt*dv1dt[idx] + av1[idx];
			if (ref_rho[idx] != 917)v2[idx] = v2_old[idx] + *dt*dv2dt[idx] + av2[idx];
			if (ref_rho[idx] != 917)x1[idx] = x1[idx] + *dt*v1[idx];
			if (ref_rho[idx] != 917)x2[idx] = x2[idx] + *dt*v2[idx];
			hsml[idx] = hsml[idx] + *dt*dhsmldt[idx];
			energy[idx] = energy_old[idx] + *dt* mass[idx] * dhdt[idx] / rho[idx];
		}
		if (ghost[idx])
		{
			rho[idx] = rho_old[idx] + *dt*drhodt[idx];
			energy[idx] = energy_old[idx] + *dt* mass[idx] * dhdt[idx] / rho[idx];
		}

	}
}

__global__ void find_extremes(int ntotal, double *hsml, double *eta, double *rho, double *v1, double *v2, double *dv1dt, double *dv2dt, double *hsml_min, double *eta_max, double *dvdt_max, double *rho_min, double *v2_max, int *fluidcode)
{

	for (int i = 1; i <= ntotal; i++)
	{
		if (i == 1)
		{
			*hsml_min = hsml[1];
			*eta_max = eta[1];
			*dvdt_max = (dv1dt[1] * dv1dt[1] + dv2dt[1] * dv2dt[1]);
			*rho_min = rho[1];
			*v2_max = (v1[1] * v1[1] + v2[1] * v2[1]);
		}
		else if (fluidcode[i] != -1)
		{
			*hsml_min = min(*hsml_min, hsml[i]);
			*eta_max = max(*eta_max, eta[i]);
			*dvdt_max = max(*dvdt_max, (dv1dt[i] * dv1dt[i] + dv2dt[i] * dv2dt[i]));
			*rho_min = min(*rho_min, rho[i]);
			*v2_max = max(*v2_max, (v1[i] * v1[i] + v2[i] * v2[i]));
		}
	}
	*dvdt_max = sqrt(*dvdt_max);

}

__global__ void timestep_calc(double c, double *hsml_min, double *eta_max, double *rho_min, double *dvdt_max, double *dt, int *reduce_timestep_flag)
{
	*dt = 100;
	*dt = min(*dt, 0.25**hsml_min / c);
	*dt = min(*dt, 0.125**hsml_min**hsml_min**rho_min / *eta_max);
	*dt = min(*dt, 0.25*sqrt(*hsml_min / *dvdt_max));
	//if (*reduce_timestep_flag)*dt /= 10;
	//printf("%.12f\n", *dt);
}

__global__ void particle_state(int ntotal, double *temp, double *energy, double *sp_heat, double *therm_cond, int *fluidcode, int* wall, int* state, int* Interface, int* ghost, double *mass, double latent_heat, double *dmgdt, int *redis_flag, double *dt)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx == 0)*redis_flag = 0;
	if (idx > 0 && idx <= ntotal)
	{

		dmgdt[idx] = 0;
		if (fluidcode[idx] == 1 && !wall[idx])
		{
			if (state[idx] == SOLID)
			{
				if (mass[idx] != 0)temp[idx] = energy[idx] / (mass[idx] * sp_heat[idx]);
				else temp[idx] = 0;
				Interface[idx] = 0;
			}
			if (state[idx] == LIQUID)
			{
				if (energy[idx] - mass[idx] * latent_heat <= 0)
				{
					dmgdt[idx] = (mass[idx] * latent_heat - energy[idx]) / latent_heat;
					temp[idx] = 0;
					mass[idx] -= dmgdt[idx];
					dmgdt[idx] = dmgdt[idx] / (*dt);
					//if (mass[idx] < 0.2*mass[1]) *redis_flag = 1;
					energy[idx] = mass[idx] * latent_heat;

				}
				else
				{
					//if (idx == 20)printf("yup");
					temp[idx] = (energy[idx] - latent_heat*mass[idx]) / (mass[idx] * sp_heat[idx]);
					//Interface[idx] = 0;
				}
			}
		}
	}
}

__global__ void derivative_sum_util(int ntotal, double *dv1dt, double *dv2dt, double *dS1dt, double *dS2dt, double *exdv1dt, double *exdv2dt)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > 0 && idx <= ntotal)
	{
		dv1dt[idx] += dS1dt[idx] + exdv1dt[idx];
		dv2dt[idx] += dS2dt[idx] + exdv2dt[idx];
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////// PRIMARY VARIABLES ASSOCIATED WITH PARTICLES
////// x1
////// x2
////// v1
////// v2
////// rho
////// rho_ref
////// pressure
////// mass
////// eta
////// hsml - smoothing length
////// temp
////// therm_cond
////// sp_heat
////// energy
////// fluid_code
////// state
////// ghost
////// ghost_state
////// wall
////// boundry
////// Interface
//////////////////////////////////////////////////////////////////////////////////////////////////////////////


void simulation()
{
	int ntotal;
	double length;
	double part_spacing;
	int timestep = 1;
	int thermal_properties_input_flag = 0;
	ifstream gridfile;
	gridfile.open("grid.txt");
	gridfile >> timestep >> length >> part_spacing >> ntotal;
	int arraySize = 5 * ntotal;
	//////////////////////////////////////////////////////////////////////////////////////////////////////
	//////// PARTICLE PROPERTIES

	double *mass_h = new double[arraySize + 1], *mass;
	double *rho_h = new double[arraySize + 1], *rho;
	double *ref_rho_h = new double[arraySize + 1], *ref_rho;
	double *eta_h = new double[arraySize + 1], *eta;
	double *pressure_h = new double[arraySize + 1], *pressure;
	double *v1_h = new double[arraySize + 1], *v1;
	double *v2_h = new double[arraySize + 1], *v2;
	double *x1_h = new double[arraySize + 1], *x1;
	double *x2_h = new double[arraySize + 1], *x2;
	double *hsml_h = new double[arraySize + 1], *hsml;
	double *temp_h = new double[arraySize + 1], *temp;
	double *therm_cond_h = new double[arraySize + 1], *therm_cond;
	double *spheat_h = new double[arraySize + 1], *spheat;
	double *energy_h = new double[arraySize + 1], *energy;
	double *dmgdt_h = new double[arraySize + 1], *dmgdt;
	double *v1_old_h = new double[arraySize + 1], *v1_old;
	double *v2_old_h = new double[arraySize + 1], *v2_old;
	double *rho_old_h = new double[arraySize + 1], *rho_old;
	double *energy_old_h = new double[arraySize + 1], *energy_old;

	//COLOR CODES
	int *master_tag_h = new int[arraySize + 1], *master_tag;
	int *fluid_code_h = new int[arraySize + 1], *fluid_code;
	int *boundry_h = new int[arraySize + 1], *boundry;
	int *wall_h = new int[arraySize + 1], *wall;
	int *Interface_h = new int[arraySize + 1], *Interface;
	int *state_h = new int[arraySize + 1], *state;
	////////////////////////////////////////////////////////////////////////////////////////////////////////

	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////// VARIABLE DEFAULTS
	for (int i = 0; i <= arraySize; i++)
	{
		master_tag_h[i] = 1;
		mass_h[i] = 0;
		rho_h[i] = 0;
		ref_rho_h[i] = 0;
		eta_h[i] = 0;
		pressure_h[i] = 0;
		v1_h[i] = 0;
		v2_h[i] = 0;
		x1_h[i] = 0;
		x2_h[i] = 0;
		hsml_h[i] = 0;
		temp_h[i] = 0;
		therm_cond_h[i] = 0;
		spheat_h[i] = 0;
		energy_h[i] = 0;
		dmgdt_h[i] = 0;
		fluid_code_h[i] = 0;
		boundry_h[i] = 0;
		wall_h[i] = 0;
		Interface_h[i] = 0;
		state_h[i] = 0;
		v1_old_h[i] = 0;
		v2_old_h[i] = 0;
		rho_old_h[i] = 0;
		energy_old_h[i] = 0;

	}

	////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////// FILE READ
	for (int i = 1; i <= ntotal; i++) {
		gridfile >> x1_h[i] >> x2_h[i] >> v1_h[i] >> v2_h[i] >> rho_h[i] >> ref_rho_h[i] >> pressure_h[i] >> mass_h[i] >> eta_h[i] >> hsml_h[i];
		gridfile >> fluid_code_h[i] >> boundry_h[i] >> wall_h[i];
	}


	// CHECK IF THERMAL PROPERTIS ARE INCLUDED IN THE FILE
	gridfile >> thermal_properties_input_flag;
	// READ THERMAL PROPERTIES
	//if (thermal_properties_input_flag)thermal_properties_input_flag = 0;
	if (thermal_properties_input_flag)
	{
		for (int i = 1; i <= ntotal; i++) {
			gridfile >> temp_h[i] >> energy_h[i] >> therm_cond_h[i] >> spheat_h[i] >> state_h[i];
		}
	}
	//thermal_properties_input_flag = 0;
	// CHECK if 'old' variable values are included in the file;
	int old_variables_flag;
	gridfile >> old_variables_flag;
	if (old_variables_flag)
	{
		string s;
		getline(gridfile, s);
		for (int i = 1; i <= ntotal; i++) {
			gridfile >> v1_old_h[i] >> v2_old_h[i] >> energy_old_h[i] >> rho_old_h[i];
		}
	}
	/////////////////////////////////////////////////////////////////////////////////////////////////////////



	/////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////// DEVICE-SIDE MEMORY ALLOCATION
	size_t size = (arraySize + 1) * sizeof(double);
	hipMalloc((void**)&x1, size);
	hipMalloc((void**)&x2, size);
	hipMalloc((void**)&v1, size);
	hipMalloc((void**)&v2, size);
	hipMalloc((void**)&rho, size);
	hipMalloc((void**)&mass, size);
	hipMalloc((void**)&pressure, size);
	hipMalloc((void**)&ref_rho, size);
	hipMalloc((void**)&eta, size);
	hipMalloc((void**)&hsml, size);
	hipMalloc((void**)&temp, size);
	hipMalloc((void**)&energy, size);
	hipMalloc((void**)&spheat, size);
	hipMalloc((void**)&therm_cond, size);
	hipMalloc((void**)&dmgdt, size);
	hipMalloc((void**)&v1_old, size);
	hipMalloc((void**)&v2_old, size);
	hipMalloc((void**)&energy_old, size);
	hipMalloc((void**)&rho_old, size);

	size = (arraySize + 1) * sizeof(int);
	hipMalloc((void**)&master_tag, size);
	hipMalloc((void**)&Interface, size);
	hipMalloc((void**)&fluid_code, size);
	hipMalloc((void**)&boundry, size);
	hipMalloc((void**)&wall, size);
	hipMalloc((void**)&state, size);

	size = (arraySize + 1) * sizeof(double);
	hipMemcpy(x1, x1_h, size, hipMemcpyHostToDevice);
	hipMemcpy(x2, x2_h, size, hipMemcpyHostToDevice);
	hipMemcpy(v1, v1_h, size, hipMemcpyHostToDevice);
	hipMemcpy(v2, v2_h, size, hipMemcpyHostToDevice);
	hipMemcpy(rho, rho_h, size, hipMemcpyHostToDevice);
	hipMemcpy(mass, mass_h, size, hipMemcpyHostToDevice);
	hipMemcpy(pressure, pressure_h, size, hipMemcpyHostToDevice);
	hipMemcpy(ref_rho, ref_rho_h, size, hipMemcpyHostToDevice);
	hipMemcpy(eta, eta_h, size, hipMemcpyHostToDevice);
	hipMemcpy(hsml, hsml_h, size, hipMemcpyHostToDevice);
	hipMemcpy(temp, temp_h, size, hipMemcpyHostToDevice);
	hipMemcpy(energy, energy_h, size, hipMemcpyHostToDevice);
	hipMemcpy(therm_cond, therm_cond_h, size, hipMemcpyHostToDevice);
	hipMemcpy(spheat, spheat_h, size, hipMemcpyHostToDevice);
	hipMemcpy(dmgdt, dmgdt_h, size, hipMemcpyHostToDevice);
	hipMemcpy(v1_old, v1_old_h, size, hipMemcpyHostToDevice);
	hipMemcpy(v2_old, v2_old_h, size, hipMemcpyHostToDevice);
	hipMemcpy(energy_old, energy_old_h, size, hipMemcpyHostToDevice);
	hipMemcpy(rho_old, rho_old_h, size, hipMemcpyHostToDevice);

	size = (arraySize + 1) * sizeof(int);
	hipMemcpy(master_tag, master_tag_h, size, hipMemcpyHostToDevice);
	hipMemcpy(Interface, Interface_h, size, hipMemcpyHostToDevice);
	hipMemcpy(fluid_code, fluid_code_h, size, hipMemcpyHostToDevice);
	hipMemcpy(boundry, boundry_h, size, hipMemcpyHostToDevice);
	hipMemcpy(wall, wall_h, size, hipMemcpyHostToDevice);
	hipMemcpy(state, state_h, size, hipMemcpyHostToDevice);
	//////////////////////////////////////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////// AUXILLIARY ARRAYS
	int *bucket_index;
	size = (arraySize + 1) * sizeof(int);
	hipMalloc((void**)&bucket_index, size);

	double *dv1dt, *dv2dt, *dS1dt, *dS2dt, *exdv1dt, *exdv2dt, *drhodt, *av1, *av2, *dhdt, *dhsmldt;
	size = (arraySize + 1) * sizeof(double);
	hipMalloc((void**)&dv1dt, size);
	hipMalloc((void**)&dv2dt, size);
	hipMalloc((void**)&dS1dt, size);
	hipMalloc((void**)&dS2dt, size);
	hipMalloc((void**)&exdv1dt, size);
	hipMalloc((void**)&exdv2dt, size);
	hipMalloc((void**)&drhodt, size);
	hipMalloc((void**)&av1, size);
	hipMalloc((void**)&av2, size);
	hipMalloc((void**)&dhdt, size);
	hipMalloc((void**)&dhsmldt, size);
	//////////////////////////////////////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////// GHOST PARTICLE VARIABLES
	int *ghost_state, *ghost_state_h = new int[arraySize + 1];
	int *ghost, *ghost_h = new int[arraySize + 1];
	double *dist = new double[arraySize + 1];
	size = (arraySize + 1) * sizeof(int);
	hipMalloc((void**)&ghost_state, size);
	hipMalloc((void**)&ghost, size);
	for (int i = 0; i < arraySize; i++)
	{
		ghost_h[i] = 0;
		ghost_state_h[i] = 0;
	}
	//////////////////////////////////////////////////////////////////////////////////////////////////////////


	//////////////////////////////////////////////////////////////////////////////////////////////////////////
	//////////////// BOUNDARY VARIABLES
	///// used to store the center coordinates and radius of the boundry circle, eliminating the need to tag boundry particles.
	pair <int, double> solid_line(GHOST_START_INDEX, 0);
	int boundary_variables_input_flag = 0;
	gridfile >> boundary_variables_input_flag;
	if (boundary_variables_input_flag)
	{
		gridfile >> solid_line.first;
		gridfile >> solid_line.second;
	}
	//set ghost state
	for (int i = 1; i <= arraySize; i++)
	{
		if (i >= GHOST_START_INDEX && i <= GHOST_END_INDEX)
		{
			if (i <= solid_line.first || i >= GHOST_END_INDEX - (solid_line.first - GHOST_START_INDEX))ghost_state_h[i] = 0;
			else ghost_state_h[i] = 1;
		}
		else ghost_state_h[i] = 0;
	}
	hipMemcpy(ghost_state, ghost_state_h, size, hipMemcpyHostToDevice);
	hipMemcpy(ghost, ghost_h, size, hipMemcpyHostToDevice);
	//////////////////////////////////////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////////////////////////////////////
	///////////// KERNEL RELATED ARRAYS
	int *neighbors, *neighbors_count, *neighbors_h = new int[(int)(max_neighbors*0.4*arraySize) + 1], *neighbors_count_h = new int[(int)(max_neighbors*0.4*arraySize) + 1];
	double *w, *dwdx, *dwdy;
	size = (max_neighbors*0.4*arraySize + 1) * sizeof(int);
	hipMalloc((void**)&neighbors, size);
	hipMalloc((void**)&neighbors_count, size);
	size = (max_neighbors*0.4*arraySize + 1) * sizeof(double);
	hipMalloc((void**)&w, size);
	hipMalloc((void**)&dwdx, size);
	hipMalloc((void**)&dwdy, size);
	//////////////////////////////////////////////////////////////////////////////////////////////////////////

	double *hsml_max, *tmp = new double[arraySize + 1]; hipMalloc((void**)&hsml_max, sizeof(double));

	int *nvoxel_length; hipMalloc((void**)&nvoxel_length, sizeof(int));


	double epsilon = 0.3;
	double alpha = 0.03;
	int block_size = 320;
	int n_blocks = ntotal / block_size + 1;
	bool density_norm = false;
	int *compression_flag, compression_flag_h[1]; hipMalloc((void**)&compression_flag, sizeof(int));
	int *evolution_flag, flag[1]; hipMalloc((void**)&evolution_flag, sizeof(int));
	int *redis_flag, redis_flag_h[1]; hipMalloc((void**)&redis_flag, sizeof(int));
	int *reduce_timestep_flag, reduce_timestep_flag_h[1]; hipMalloc((void**)&reduce_timestep_flag, sizeof(int));
	reduce_timestep_flag_h[0] = 0; hipMemcpy(reduce_timestep_flag, reduce_timestep_flag_h, sizeof(int), hipMemcpyHostToDevice);
	int count_timestep = 0;
	int *new_ntotal, new_ntotal_h[1]; hipMalloc((void**)&new_ntotal, sizeof(int));
	int old_ntotal = ntotal;
	int latent_heat = 80 * 4180 / 7;
	double total_mass = 0, initial_total_mass;
	double liquid_mass_old = 0;

	double *hsml_min, *eta_max, *dvdt_max, *rho_min, *dt, *v2_max;
	double *dt_h = new double[1];
	double *v2max_h = new double[1];
	hipMalloc((void**)&hsml_min, sizeof(double));
	hipMalloc((void**)&eta_max, sizeof(double));
	hipMalloc((void**)&rho_min, sizeof(double));
	hipMalloc((void**)&dvdt_max, sizeof(double));
	hipMalloc((void**)&dt, sizeof(double));
	hipMalloc((void**)&v2_max, sizeof(double));

	//////////////////////////////////////////////////////////////////////////////////////
	/////// RENDER RELATED ARRAYS
	double **tmp_render = new double*[3]; for (int i = 0; i < 3; i++) tmp_render[i] = new double[arraySize + 1];
	vector <int> marker_vector;
	//////////////////////////////////////////////////////////////////////////////////////

	double st, en, totTime = 0;

	int max_timestep = 130000;
	double time_elapsed = 0;
	if (timestep > max_timestep)max_timestep = timestep;

	//Get total mass of liquid+solid+ghost particles;
	initial_total_mass = initial_total_mass_util(ntotal, mass_h, wall_h);
	bool recompute_kernel = true;
	cout << "total mass: " << 1000 * initial_total_mass << endl;



	//*************************************************************************************
	// The primary simulation loop
	//*************************************************************************************
	while (timestep <= max_timestep)
	{


		st = clock();
		*flag = 0;
		n_blocks = ntotal / block_size + 1;

		if (timestep != 1) leapfrog_part_1 << <n_blocks, block_size >> >(ntotal, v1, v2, mass, ref_rho, energy, dhdt, v1_old, v2_old, rho, rho_old, energy_old, wall, fluid_code, ghost, drhodt, dv1dt, dv2dt, dt);
		max_hsml << <1, 1 >> >(ntotal, hsml, hsml_max, fluid_code);

		if (recompute_kernel)
		{
			index_particles << <n_blocks, block_size >> >(ntotal, bucket_index, x1, x2, length, kern_constant, hsml_max, nvoxel_length);
			compute_kernel << <n_blocks, block_size >> >(ntotal, bucket_index, nvoxel_length, x1, x2, hsml, neighbors, neighbors_count, w, dwdx, dwdy, ghost, ghost_state, wall, fluid_code, Interface, kern_constant, part_spacing, max_neighbors);
		}

		equation_of_state << <n_blocks, block_size >> >(ntotal, pressure, rho, ref_rho, fluid_code, Interface, p0, gamma, back_press, c);



		n_blocks = 8 * ntotal / block_size + 1;
		compute_derivatives << <n_blocks, block_size >> >(ntotal, x1, x2, v1, v2, rho, mass, pressure, hsml, neighbors, neighbors_count, w, dwdx, dwdy, eta, therm_cond, temp, dv1dt, dv2dt, dS1dt, dS2dt, exdv1dt, exdv2dt, av1, av2, drhodt, dhdt, dmgdt, dhsmldt, wall, fluid_code, Interface, state, ghost_state, ghost, length, epsilon, gamma, c, back_press, max_neighbors, kern_constant, constant_gravity, S_ll, S_ls, part_spacing);
		n_blocks = ntotal / block_size + 1;

		//force_comparison_util(dv1dt, dv2dt,dS1dt, dS2dt, exdv1dt, exdv2dt, ntotal);

		derivative_sum_util << <n_blocks, block_size >> >(ntotal, dv1dt, dv2dt, dS1dt, dS2dt, exdv1dt, exdv2dt);

		find_extremes << <1, 1 >> >(ntotal, hsml, eta, rho, v1, v2, dv1dt, dv2dt, hsml_min, eta_max, dvdt_max, rho_min, v2_max, fluid_code);

		timestep_calc << <1, 1 >> >(c, hsml_min, eta_max, rho_min, dvdt_max, dt, reduce_timestep_flag);
		hipMemcpy(dt_h, dt, sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(v2max_h, v2_max, sizeof(double), hipMemcpyDeviceToHost);
		if (ref_velocity*ref_velocity < *v2max_h)
		{
			cout << "Particle velocity exceeded reference velocity: " << sqrt(*v2max_h) << endl;
			getchar();
		}
		time_elapsed += dt_h[0];
		if (timestep == 1 && !old_variables_flag) leapfrog_part_1 << <n_blocks, block_size >> >(ntotal, v1, v2, mass, ref_rho, energy, dhdt, v1_old, v2_old, rho, rho_old, energy_old, wall, fluid_code, ghost, drhodt, dv1dt, dv2dt, dt);
		else leapfrog_part_2 << <n_blocks, block_size >> >(ntotal, v1, v2, mass, ref_rho, temp, spheat, energy, dhdt, v1_old, v2_old, av1, av2, x1, x2, rho, rho_old, energy_old, hsml, wall, fluid_code, ghost, state, drhodt, dv1dt, dv2dt, dhsmldt, dt, compression_flag);
		hipMemcpy(compression_flag_h, compression_flag, sizeof(int), hipMemcpyDeviceToHost);

		if (*compression_flag_h)
		{
			cout << "particles density deviation beyond 3%" << endl;
			//getchar();
		}

		//if(timestep >300 && timestep<1000) temp_move_ghost_util << <n_blocks, block_size >> >(ntotal, x2, v2,mass, part_spacing, dt);

		hipDeviceSynchronize();





	

		hipMemcpy(x1_h, x1, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
		hipMemcpy(x2_h, x2, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);

		en = clock();
		render_scene(ntotal, x1_h, x2_h, mass_h, fluid_code_h, wall_h, state_h, ghost_state_h, ghost_h, tmp_render, marker_vector, part_spacing);
		//render_scene_mass(ntotal, x1_h, x2_h, mass_h, wall_h, state_h, ghost_h, tmp_render, part_spacing);
		time_elapsed += *dt_h;

		cout << "timestep no: " << timestep << " timestep length: " << *dt_h << " total time elapsed: " << time_elapsed << endl;
		//cout << "Time elapsed: " << time_elapsed << endl;
		//render_scene_mass(ntotal, x1_h, x2_h, mass_h, wall_h, state_h, ghost_h, tmp_render, length, 40, part_spacing);
		cout << "timestep no: " << timestep << "      Computation time: " << en - st << endl;
		//cout << "ntotal: " << ntotal << "old ntotal: " << old_ntotal <<  endl;
		//cout << "mass ratio at 800: : " << mass_h[865]/(917*part_spacing*part_spacing) << endl;
		//cout << "mass balance (should be 1): " << total_mass / initial_total_mass << endl;
		//cout << "liquid mass/old mass: ";


		//Dump data every hundred steps
		if (timestep % 1000 == 0 || timestep == 50998)
		{

			cout << "DUMPING DATA....." << endl;
			dump_data(ntotal, x1, x2, v1, v2, rho, ref_rho, pressure, mass, eta, hsml, temp, energy, therm_cond, spheat, fluid_code, state, ghost, ghost_state, wall, boundry, Interface, v1_old, v2_old, energy_old, rho_old, x1_h, x2_h, v1_h, v2_h, rho_h, ref_rho_h, pressure_h, mass_h, eta_h, hsml_h, temp_h, energy_h, therm_cond_h, spheat_h, fluid_code_h, state_h, ghost_h, ghost_state_h, wall_h, boundry_h, Interface_h, v1_old_h, v2_old_h, energy_old_h, rho_old_h, arraySize, length, part_spacing, timestep, solid_line);

		}
		timestep++;
		//getchar();
		marker_vector.clear();
		if (timestep > max_timestep)
		{
			cout << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl << endl;
			cout << "Max timestep reached. Enter the number of extra iterations to perform:" << endl;
			int n; cin >> n;
			max_timestep += n;
		}
	}



}

////////////////////////////////////////////////////////////////////////////////////////////
//////  RENDER SCENE
int render_scene(int ntotal, double *x1, double *x2, double *mass, int *fluid_code, int *wall, int *state, int *test_array, int *ghost, double** temp, vector <int> &marker_vector, double part_spacing)
{
	int i;
	for (i = 1; i <= ntotal; i++)
	{
		//temp[1][i] = (x1[i] + 5 * length / nPartSide) / (length + 10 * length / nPartSide);
		//temp[2][i] = (x2[i] + 5 * length / nPartSide) / (0.7*length + 3 * length / nPartSide);
		temp[1][i] = x1[i] / RENDER_X_SCALE;
		temp[2][i] = x2[i] / RENDER_Y_SCALE;
		temp[1][i] = 2 * (temp[1][i] + RENDER_X_OFFSET);
		temp[2][i] = 2 * (temp[2][i] + RENDER_Y_OFFSET);
	}

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT); //this was taken from the render function

	glBegin(GL_TRIANGLES);


	for (i = 1; i <= ntotal; i++)
	{
		if (true)glColor3f(1, 1, 1);
		if (fluid_code[i] == 1)glColor3f(0, 1, 0);

		if (state[i] == 0)glColor3f(0, 0, 1);
		if (wall[i])glColor3f(1, 0, 0);
		if (ghost[i])glColor3f(1, 1, 1);
		//if (i > ntotal/5) glColor3f(1, 0, 0);
		//if (test_array[i])glColor3f(1, 0.647059, 0);
		//if (i == 357)glColor3f(1, 0, 0);
		if (fluid_code[i] == -1)glColor3f(0, 0, 0);

		if (i)
		{

			glVertex3f(temp[1][i], temp[2][i] + 0.004, 0.0);
			glVertex3f(temp[1][i] - 0.004, temp[2][i] - 0.004, 0.0);
			glVertex3f(temp[1][i] + 0.004, temp[2][i] - 0.004, 0.0);
		}



	}
	for (int j = 0; j < marker_vector.size(); j++)
	{
		i = marker_vector[j];
		glColor3f(1, 0, 0);
		glVertex3f(temp[1][i], temp[2][i] + 0.004, 0.0);
		glVertex3f(temp[1][i] - 0.004, temp[2][i] - 0.004, 0.0);
		glVertex3f(temp[1][i] + 0.004, temp[2][i] - 0.004, 0.0);
	}
	glEnd();

	glutSwapBuffers();

	return 0;
}
////////////////////////////////////////////////////////////////////////////////////////////
int render_scene_mass(int ntotal, double *x1, double *x2, double* mass, int *wall, int *state, int *ghost, double** temp, double part_spacing)
{
	int i;
	for (i = 1; i <= ntotal; i++)
	{
		temp[1][i] = x1[i] / RENDER_X_SCALE;
		temp[2][i] = x2[i] / RENDER_Y_SCALE;
		temp[1][i] = 2 * (temp[1][i] + RENDER_X_OFFSET);
		temp[2][i] = 2 * (temp[2][i] + RENDER_Y_OFFSET);
	}
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glLineWidth(2.5);
	glColor3f(1, 1, 1);
	glBegin(GL_LINES);
	for (i = 1; i <= ntotal; i++)
	{
		if (ghost[i])
		{

			glVertex3f(temp[1][i], 0.0, 0.0);
			glVertex3f(temp[1][i], mass[i] / (917 * part_spacing*part_spacing), 0.0);
		}
	}
	glEnd();

	glutSwapBuffers();
	return 0;
}
////////////////////////////////////////////////////////////////////////////////////////////
///// DUMP DATA
void dump_data(int ntotal, double *x1, double *x2, double *v1, double *v2, double *rho, double *ref_rho, double *pressure, double *mass, double *eta, double *hsml, double *temp, double *energy, double *therm_cond, double *sp_heat, int *fluid_code, int *state, int *ghost, int* ghost_state, int* wall, int* boundry, int* Interface, double *v1_old, double *v2_old, double *energy_old, double *rho_old, double *x1_h, double *x2_h, double *v1_h, double *v2_h, double *rho_h, double *ref_rho_h, double *pressure_h, double *mass_h, double *eta_h, double *hsml_h, double *temp_h, double *energy_h, double *therm_cond_h, double *sp_heat_h, int *fluid_code_h, int *state_h, int *ghost_h, int* ghost_state_h, int* wall_h, int* boundry_h, int* Interface_h, double *v1_old_h, double *v2_old_h, double *energy_old_h, double *rho_old_h, int arraySize, double length, double part_spacing, int index, pair <int, double> solid_line)
{
	hipMemcpy(x1_h, x1, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(x2_h, x2, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(v1_h, v1, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(v2_h, v2, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(rho_h, rho, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(ref_rho_h, ref_rho, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(pressure_h, pressure, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(mass_h, mass, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(eta_h, eta, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(hsml_h, hsml, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(temp_h, temp, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(energy_h, energy, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(therm_cond_h, therm_cond, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(sp_heat_h, sp_heat, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);

	hipMemcpy(fluid_code_h, fluid_code, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(state_h, state, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(ghost_h, ghost, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(ghost_state_h, ghost_state, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(wall_h, wall, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(boundry_h, boundry, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);
	hipMemcpy(Interface_h, Interface, ((arraySize + 1) * sizeof(int)), hipMemcpyDeviceToHost);

	hipMemcpy(v1_old_h, v1_old, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(v2_old_h, v2_old, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(energy_old_h, energy_old, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(rho_old_h, rho_old, (arraySize + 1) * sizeof(double), hipMemcpyDeviceToHost);

	ofstream pos;
	pos.open("dump_" + to_string(index) + ".txt", 'w');
	pos << index + 1 << endl << length << endl << part_spacing << endl << ntotal << endl;

	for (int i = 1; i <= ntotal; i++)
	{
		pos << x1_h[i] << ' ' << x2_h[i] << ' ' << v1_h[i] << ' ' << v2_h[i] << ' ' << rho_h[i] << ' ' << ref_rho_h[i] << ' ' << pressure_h[i] << ' ' << mass_h[i] << ' ' << eta_h[i] << ' ' << hsml_h[i];
		pos << ' ' << fluid_code_h[i] << ' ' << boundry_h[i] << ' ' << wall_h[i] << endl;
	}
	pos << '1' << endl;
	for (int i = 1; i <= ntotal; i++)
	{
		pos << temp_h[i] << ' ' << energy_h[i] << ' ' << therm_cond_h[i] << ' ' << sp_heat_h[i] << ' ' << state_h[i] << endl;
	}
	pos << '1' << endl;
	for (int i = 1; i <= ntotal; i++)
	{
		pos << v1_old_h[i] << ' ' << v2_old_h[i] << ' ' << energy_old_h[i] << ' ' << rho_old_h[i] << endl;
	}
	pos << '1' << endl;
	pos << solid_line.first << ' ' << solid_line.second << endl;
	//Uncomment below to take screenshots. Needs FreeImage Library.
	/*BYTE* pixels = new BYTE[3 * 1000 * 1000];

	glReadPixels(0, 0, 1000, 1000, GL_RGB, GL_UNSIGNED_BYTE, pixels);

	// Convert to FreeImage format & save to file
	FIBITMAP* image = FreeImage_ConvertFromRawBits(pixels, 1000, 1000, 3 * 1000, 24, 0x0000FF, 0xFF0000, 0x00FF00, false);
	string name = to_string(index) + ".bmp";
	char *a = new char[name.size() + 1];
	a[name.size()] = 0;
	memcpy(a, name.c_str(), name.size());

	FreeImage_Save(FIF_BMP, image, a, 0);

	// Free resources
	FreeImage_Unload(image);
	delete[] pixels;*/
	pos.close();
}
///////////////////////////////////////////////////////////////////////////////////////////
////// TOTAL MASS CALCULATOR
double initial_total_mass_util(int ntotal, double *mass_h, int *wall_h)
{
	double tot = 0;
	for (int i = 1; i <= ntotal; i++)
	{
		if (!wall_h[i])tot += mass_h[i];
	}
	return tot;
}
///////////////////////////////////////////////////////////////////////////////////////////
//////ARRAY DUMP FUNCTION
void write_array(int *arr, int len, string name)
{
	int size = len * sizeof(int);
	int *host_array = new int[len];
	hipMemcpy(host_array, arr, size, hipMemcpyDeviceToHost);
	ofstream pos;
	pos.open(name + ".txt");
	for (int i = 0; i < len; i++)
	{
		pos << host_array[i] << endl;
	}
}
void write_array(double *arr, int len, string name)
{
	int size = len * sizeof(double);
	double *host_array = new double[len];
	hipMemcpy(host_array, arr, size, hipMemcpyDeviceToHost);
	ofstream pos;
	pos.open(name + ".txt");
	for (int i = 0; i < len; i++)
	{
		pos << host_array[i] / 7.20269e-7 << endl;
	}

	delete[] host_array;
}
///////////////////////////////////////////////////////////////////////////////////////////
////// PARTICLE FORCE COMPARISON UTIL
void force_comparison_util(double *dv1dt, double *dv2dt, double *dS1dt, double *dS2dt, double *exdv1dt, double *exdv2dt, int len)
{
	int size = len * sizeof(double);
	double *dv1dt_h = new double[len + 1];
	double *dv2dt_h = new double[len + 1];
	double *dS1dt_h = new double[len + 1];
	double *dS2dt_h = new double[len + 1];
	double *exdv1dt_h = new double[len + 1];
	double *exdv2dt_h = new double[len + 1];
	hipMemcpy(dv1dt_h, dv1dt, size, hipMemcpyDeviceToHost);
	hipMemcpy(dv2dt_h, dv2dt, size, hipMemcpyDeviceToHost);
	hipMemcpy(dS1dt_h, dS1dt, size, hipMemcpyDeviceToHost);
	hipMemcpy(dS2dt_h, dS2dt, size, hipMemcpyDeviceToHost);
	hipMemcpy(exdv2dt_h, exdv2dt, size, hipMemcpyDeviceToHost);
	hipMemcpy(exdv1dt_h, exdv1dt, size, hipMemcpyDeviceToHost);

	ofstream pos;
	pos.open("force_comparison.txt");
	pos << "pressure - surface tension - external force - gravity" << endl;
	for (int i = 1; i <= len; i++)
	{
		pos << sqrt(pow(dv1dt_h[i], 2) + pow(dv2dt_h[i] + constant_gravity, 2)) / constant_gravity << "  " << sqrt(pow(dS1dt_h[i], 2) + pow(dS2dt_h[i], 2)) / constant_gravity << ' ' << sqrt(pow(exdv1dt_h[i], 2) + pow(exdv2dt_h[i], 2)) / constant_gravity << "  1" << endl;
	}
	pos.close();
	delete[] dv1dt_h, dv2dt_h, dS1dt_h, dS2dt_h, exdv1dt_h, exdv2dt_h;
}
